#include "hip/hip_runtime.h"
/*
 MD.c - a simple molecular dynamics program for simulating real gas properties of Lennard-Jones particles.
 
 Copyright (C) 2016  Jonathan J. Foley IV, Chelsea Sweet, Oyewumi Akinfenwa
 
 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License as published by
 the Free Software Foundation, either version 3 of the License, or
 (at your option) any later version.
 
 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.
 
 You should have received a copy of the GNU General Public License
 along with this program.  If not, see <http://www.gnu.org/licenses/>.
 
 Electronic Contact:  foleyj10@wpunj.edu
 Mail Contact:   Prof. Jonathan Foley
 Department of Chemistry, William Paterson University
 300 Pompton Road
 Wayne NJ 07470
 
 */
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>
#include<unistd.h>
#include"MD.h"
#include<hip/hip_runtime.h>

// Number of particles
const int N=5000;

#define NUM_THREADS_PER_BLOCK 128
#define NUM_BLOCKS ((N + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK)
#define MAXPART NUM_BLOCKS*NUM_THREADS_PER_BLOCK

double *da, *dr, *dpot, pot[NUM_BLOCKS];
int bytes = N * 3 * sizeof(double);

//  Lennard-Jones parameters in natural units!
double m = 1.0;
double kB = 1.0;


double NA = 6.022140857e23;
double kBSI = 1.38064852e-23;  // m^2*kg/(s^2*K)

//  Size of box, which will be specified in natural units
double L;

//  Initial Temperature in Natural Units
double Tinit;  //2;
//  Vectors!
//  Position
double r[3*N];
//  Velocity
double v[3*N];
//  Acceleration
double a[3*N];

double PE = 0;
double mvs = 0;
double KE = 0;

// atom type
char atype[10];
//  Function prototypes
//  initialize positions on simple cubic lattice, also calls function to initialize velocities
void initialize();  
//  update positions and velocities using Velocity Verlet algorithm 
//  print particle coordinates to file for rendering via VMD or other animation software
//  return 'instantaneous pressure'
double VelocityVerlet(double dt, int iter, FILE *fp);  
//  Compute Force using F = -dV/dr
//  solve F = ma for use in Velocity Verlet
//  Compute total potential energy from particle coordinates
void computeAccelsAndPotential();
//  Compute Force using F = -dV/dr
//  Numerical Recipes function for generation gaussian distribution
double gaussdist();
//  Initialize velocities according to user-supplied initial Temperature (Tinit)
void initializeVelocities();
//  Compute mean squared velocity from particle velocities and total kinetic energy from particle mass and velocities
void MeanSqdVelocityAndKinetic();

int main()
{
    
    //  variable delcarations
    int i;
    double dt, Vol, Temp, Press, Pavg, Tavg, rho;
    double VolFac, TempFac, PressFac, timefac;
    double gc, Z;
    char prefix[1000], tfn[1000], ofn[1000], afn[1000];
    FILE *tfp, *ofp, *afp;
    
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("                  WELCOME TO WILLY P CHEM MD!\n");
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n  ENTER A TITLE FOR YOUR CALCULATION!\n");
    scanf("%s",prefix);
    strcpy(tfn,prefix);
    strcat(tfn,"_traj.xyz");
    strcpy(ofn,prefix);
    strcat(ofn,"_output.txt");
    strcpy(afn,prefix);
    strcat(afn,"_average.txt");
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("                  TITLE ENTERED AS '%s'\n",prefix);
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    
    /*     Table of values for Argon relating natural units to SI units:
     *     These are derived from Lennard-Jones parameters from the article
     *     "Liquid argon: Monte carlo and molecular dynamics calculations"
     *     J.A. Barker , R.A. Fisher & R.O. Watts
     *     Mol. Phys., Vol. 21, 657-673 (1971)
     *
     *     mass:     6.633e-26 kg          = one natural unit of mass for argon, by definition
     *     energy:   1.96183e-21 J      = one natural unit of energy for argon, directly from L-J parameters
     *     length:   3.3605e-10  m         = one natural unit of length for argon, directly from L-J parameters
     *     volume:   3.79499-29 m^3        = one natural unit of volume for argon, by length^3
     *     time:     1.951e-12 s           = one natural unit of time for argon, by length*sqrt(mass/energy)
     ***************************************************************************************/
    
    //  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //  Edit these factors to be computed in terms of basic properties in natural units of
    //  the gas being simulated
    
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("  WHICH NOBLE GAS WOULD YOU LIKE TO SIMULATE? (DEFAULT IS ARGON)\n");
    printf("\n  FOR HELIUM,  TYPE 'He' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR NEON,    TYPE 'Ne' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR ARGON,   TYPE 'Ar' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR KRYPTON, TYPE 'Kr' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR XENON,   TYPE 'Xe' THEN PRESS 'return' TO CONTINUE\n");
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    scanf("%s",atype);
    
    if (strcmp(atype,"He")==0) {
        
        VolFac = 1.8399744000000005e-29;
        PressFac = 8152287.336171632;
        TempFac = 10.864459551225972;
        timefac = 1.7572698825166272e-12;
        
    }
    else if (strcmp(atype,"Ne")==0) {
        
        VolFac = 2.0570823999999997e-29;
        PressFac = 27223022.27659913;
        TempFac = 40.560648991243625;
        timefac = 2.1192341945685407e-12;
        
    }
    else if (strcmp(atype,"Ar")==0) {
        
        VolFac = 3.7949992920124995e-29;
        PressFac = 51695201.06691862;
        TempFac = 142.0950000000000;
        timefac = 2.09618e-12;
        //strcpy(atype,"Ar");
        
    }
    else if (strcmp(atype,"Kr")==0) {
        
        VolFac = 4.5882712000000004e-29;
        PressFac = 59935428.40275003;
        TempFac = 199.1817584391428;
        timefac = 8.051563913585078e-13;
        
    }
    else if (strcmp(atype,"Xe")==0) {
        
        VolFac = 5.4872e-29;
        PressFac = 70527773.72794868;
        TempFac = 280.30305642163006;
        timefac = 9.018957925790732e-13;
        
    }
    else {
        
        VolFac = 3.7949992920124995e-29;
        PressFac = 51695201.06691862;
        TempFac = 142.0950000000000;
        timefac = 2.09618e-12;
        strcpy(atype,"Ar");
        
    }
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n                     YOU ARE SIMULATING %s GAS! \n",atype);
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n  YOU WILL NOW ENTER A FEW SIMULATION PARAMETERS\n");
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n\n  ENTER THE INTIAL TEMPERATURE OF YOUR GAS IN KELVIN\n");
    scanf("%lf",&Tinit);
    // Make sure temperature is a positive number!
    if (Tinit<0.) {
        printf("\n  !!!!! ABSOLUTE TEMPERATURE MUST BE A POSITIVE NUMBER!  PLEASE TRY AGAIN WITH A POSITIVE TEMPERATURE!!!\n");
        exit(0);
    }
    // Convert initial temperature from kelvin to natural units
    Tinit /= TempFac;
    
    
    printf("\n\n  ENTER THE NUMBER DENSITY IN moles/m^3\n");
    printf("  FOR REFERENCE, NUMBER DENSITY OF AN IDEAL GAS AT STP IS ABOUT 40 moles/m^3\n");
    printf("  NUMBER DENSITY OF LIQUID ARGON AT 1 ATM AND 87 K IS ABOUT 35000 moles/m^3\n");
    
    scanf("%lf",&rho);

    Vol = N/(rho*NA);
    
    Vol /= VolFac;
    
    //  Limiting N to MAXPART for practical reasons
    if (N>=MAXPART) {
        
        printf("\n\n\n  MAXIMUM NUMBER OF PARTICLES IS %i\n\n  PLEASE ADJUST YOUR INPUT FILE ACCORDINGLY \n\n", MAXPART);
        exit(0);
        
    }
    //  Check to see if the volume makes sense - is it too small?
    //  Remember VDW radius of the particles is 1 natural unit of length
    //  and volume = L*L*L, so if V = N*L*L*L = N, then all the particles
    //  will be initialized with an interparticle separation equal to 2xVDW radius
    if (Vol<N) {
        
        printf("\n\n\n  YOUR DENSITY IS VERY HIGH!\n\n");
        printf("  THE NUMBER OF PARTICLES IS %i AND THE AVAILABLE VOLUME IS %f NATURAL UNITS\n",N,Vol);
        printf("  SIMULATIONS WITH DENSITY GREATER THAN 1 PARTCICLE/(1 Natural Unit of Volume) MAY DIVERGE\n");
        printf("  PLEASE ADJUST YOUR INPUT FILE ACCORDINGLY AND RETRY\n\n");
        exit(0);
    }
    // Vol = L*L*L;
    // Length of the box in natural units:
    L = pow(Vol,(1./3));
    
    //  Files that we can write different quantities to
    tfp = fopen(tfn,"w");     //  The MD trajectory, coordinates of every particle at each timestep
    ofp = fopen(ofn,"w");     //  Output of other quantities (T, P, gc, etc) at every timestep
    afp = fopen(afn,"w");    //  Average T, P, gc, etc from the simulation
    
    int NumTime;
    if (strcmp(atype,"He")==0) {
        
        // dt in natural units of time s.t. in SI it is 5 f.s. for all other gasses
        dt = 0.2e-14/timefac;
        //  We will run the simulation for NumTime timesteps.
        //  The total time will be NumTime*dt in natural units
        //  And NumTime*dt multiplied by the appropriate conversion factor for time in seconds
        NumTime=50000;
    }
    else {
        dt = 0.5e-14/timefac;
        NumTime=200;
        
    }

    hipMalloc((void **)&dr, bytes);
    hipMalloc((void **)&dpot, NUM_BLOCKS*sizeof(double));
    hipMalloc((void **)&da, bytes);
    checkCUDAError("mem allocation");
    
    //  Put all the atoms in simple crystal lattice and give them random velocities
    //  that corresponds to the initial temperature we have specified
    initialize();
    
    //  Based on their positions, calculate the ininial intermolecular forces
    //  The accellerations of each particle will be defined from the forces and their
    //  mass, and this will allow us to update their positions via Newton's law
    computeAccelsAndPotential();
    
    // Print number of particles to the trajectory file
    fprintf(tfp,"%i\n",N);
    
    //  We want to calculate the average Temperature and Pressure for the simulation
    //  The variables need to be set to zero initially
    Pavg = 0;
    Tavg = 0;
    
    
    int tenp = floor(NumTime/10);
    double Volume = Vol * VolFac;
    fprintf(ofp,"  time (s)              T(t) (K)              P(t) (Pa)           Kinetic En. (n.u.)     Potential En. (n.u.) Total En. (n.u.)\n");
    printf("  PERCENTAGE OF CALCULATION COMPLETE:\n  [");
    for (i=0; i<NumTime+1; i++) {
        
        //  This just prints updates on progress of the calculation for the users convenience
        if (i==tenp) printf(" 10 |");
        else if (i==2*tenp) printf(" 20 |");
        else if (i==3*tenp) printf(" 30 |");
        else if (i==4*tenp) printf(" 40 |");
        else if (i==5*tenp) printf(" 50 |");
        else if (i==6*tenp) printf(" 60 |");
        else if (i==7*tenp) printf(" 70 |");
        else if (i==8*tenp) printf(" 80 |");
        else if (i==9*tenp) printf(" 90 |");
        else if (i==10*tenp) printf(" 100 ]\n");
        fflush(stdout);
        
        
        // This updates the positions and velocities using Newton's Laws
        // Also computes the Pressure as the sum of momentum changes from wall collisions / timestep
        // which is a Kinetic Theory of gasses concept of Pressure
        Press = VelocityVerlet(dt, i+1, tfp);
        Press *= PressFac;
        
        //  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
        //  Now we would like to calculate somethings about the system:
        //  Instantaneous mean velocity squared, Temperature, Pressure and Kinetic Energy
        //  We would also like to use the IGL to try to see if we can extract the gas constant
        MeanSqdVelocityAndKinetic();
        
        // Temperature from Kinetic Theory
        Temp = m*mvs/(3*kB) * TempFac;
        
        // Instantaneous gas constant and compressibility - not well defined because
        // pressure may be zero in some instances because there will be zero wall collisions,
        // pressure may be very high in some instances because there will be a number of collisions
        gc = NA*Press*Volume/(N*Temp);
        Z  = Press*Volume/(N*kBSI*Temp);
        
        Tavg += Temp;
        Pavg += Press;

        fprintf(ofp,"  %8.4e  %20.8f  %20.8f %20.8f  %20.8f  %20.8f \n",i*dt*timefac,Temp,Press,KE, PE, KE+PE);
    }
    
    // Because we have calculated the instantaneous temperature and pressure,
    // we can take the average over the whole simulation here
    Pavg /= NumTime;
    Tavg /= NumTime;
    Z = Pavg*Volume/(N*kBSI*Tavg);
    gc = NA*Pavg*Volume/(N*Tavg);
    fprintf(afp,"  Total Time (s)      T (K)               P (Pa)      PV/nT (J/(mol K))         Z           V (m^3)              N\n");
    fprintf(afp," --------------   -----------        ---------------   --------------   ---------------   ------------   -----------\n");
    fprintf(afp,"  %8.12e  %15.12f       %15.12f     %10.12f       %10.12f        %10.12e         %i\n",i*dt*timefac,Tavg,Pavg,gc,Z,Vol*VolFac,N);
    
    printf("\n  TO ANIMATE YOUR SIMULATION, OPEN THE FILE \n  '%s' WITH VMD AFTER THE SIMULATION COMPLETES\n",tfn);
    printf("\n  TO ANALYZE INSTANTANEOUS DATA ABOUT YOUR MOLECULE, OPEN THE FILE \n  '%s' WITH YOUR FAVORITE TEXT EDITOR OR IMPORT THE DATA INTO EXCEL\n",ofn);
    printf("\n  THE FOLLOWING THERMODYNAMIC AVERAGES WILL BE COMPUTED AND WRITTEN TO THE FILE  \n  '%s':\n",afn);
    printf("\n  AVERAGE TEMPERATURE (K):                 %15.12f\n",Tavg);
    printf("\n  AVERAGE PRESSURE  (Pa):                  %15.12f\n",Pavg);
    printf("\n  PV/nT (J * mol^-1 K^-1):                 %15.12f\n",gc);
    printf("\n  PERCENT ERROR of pV/nT AND GAS CONSTANT: %15.12f\n",100*fabs(gc-8.3144598)/8.3144598);
    printf("\n  THE COMPRESSIBILITY (unitless):          %15.12f \n",Z);
    printf("\n  TOTAL VOLUME (m^3):                      %10.12e \n",Vol*VolFac);
    printf("\n  NUMBER OF PARTICLES (unitless):          %i \n", N);
    
    
    
    fclose(tfp);
    fclose(ofp);
    fclose(afp);

    hipFree(dr);
    hipFree(da);
    hipFree(dpot);
    checkCUDAError("mem free");   

    return 0;
}


void initialize() {
    int n, p;
    double pos;
    
    // Number of atoms in each direction
    n = int(ceil(pow(N, 1.0/3)));

    //  spacing between atoms along a given direction
    pos = L / n;
    
    //  index for number of particles assigned positions
    p = 0;

    //  initialize positions
    for (int i=0; i<n; i++) {
        for (int j=0; j<n; j++) {
            for (int k=0; k<n; k++) {
                if (p<N) {
                    r[p*3] = (i + 0.5)*pos;
                    r[p*3 + 1] = (j + 0.5)*pos;
                    r[p*3 + 2] = (k + 0.5)*pos;
                }
                p++;
            }
        }
    }
    
    // Call function to initialize velocities
    initializeVelocities();    
}   


//  Function to calculate the averaged velocity squared and kinetic energy of the system
void MeanSqdVelocityAndKinetic() { 
    double vSquared = 0.;
    
    for (int i=0; i<N; i++) {
        double vX = v[i*3];
        double vY = v[i*3 + 1];
        double vZ = v[i*3 + 2];

        vSquared += vX*vX + vY*vY + vZ*vZ;
    }
    
    mvs = vSquared/N;
    KE = (m/2.)*vSquared;
}


__global__ void computeAccelsAndPotentialKernel(double *da, double *dr, double *dpot) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    if (idx >= N) return; // idx >= N-1

    __shared__ double sdata[NUM_THREADS_PER_BLOCK];
    sdata[tid] = 0;
    __syncthreads();

    double accelAcc[3] = {0,0,0};
    double drx = dr[idx*3], dry = dr[idx*3 + 1], drz = dr[idx*3 + 2];
    for(int j = 0; j < N; j++) {
        if(idx != j){
            double rij[3];
            rij[0] = drx - dr[j*3];
            rij[1] = dry - dr[j*3 + 1];
            rij[2] = drz - dr[j*3 + 2];

            double rSqd = rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2];
            double rSqdInv = 1.0 / rSqd;
            double rSqd3Inv = rSqdInv * rSqdInv * rSqdInv;
            double rSqd4Inv = rSqdInv * rSqd3Inv;

            sdata[tid] += 4 * rSqd3Inv * (rSqd3Inv - 1);

            // From derivative of Lennard-Jones with sigma and epsilon set equal to 1
            double f = rSqd4Inv * (48 * rSqd3Inv - 24);

            for(int k = 0; k < 3; k++){
                accelAcc[k] += rij[k] * f;
            }
        }
    }

    for(int k = 0; k < 3; k++){
        da[idx*3 + k] = accelAcc[k];
    }

    __syncthreads(); 

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }

        __syncthreads();
    }

    if (tid == 0) dpot[blockIdx.x] = sdata[0];
}

//   Uses the derivative of the Lennard-Jones potential to calculate
//   the forces on each atom.  Then uses a = F/m to calculate the
//   accelleration of each atom.
void computeAccelsAndPotential() {
    hipMemcpy(dr, r, bytes, hipMemcpyHostToDevice);
    checkCUDAError("memcpy h->d");

    // Lançamento do kernel
    computeAccelsAndPotentialKernel <<< NUM_BLOCKS, NUM_THREADS_PER_BLOCK >>> (da, dr, dpot);
    checkCUDAError("kernel invocation");

    hipMemcpy(pot, dpot, NUM_BLOCKS*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(a, da, bytes, hipMemcpyDeviceToHost);
    checkCUDAError("memcpy d->h");

    double potAccum = 0;
    for(int i = 0; i < NUM_BLOCKS; i++){
        potAccum += pot[i];
    }

    PE = potAccum;
}


// returns sum of dv/dt*m/A (aka Pressure) from elastic collisions with walls
double VelocityVerlet(double dt, int iter, FILE *fp) {
    double psum = 0., halfDT = 0.5*dt;
    
    //  Update positions and velocity with current velocity and acceleration
    for (int i=0; i<N; i++) {
        r[i*3] += dt*(v[i*3] + halfDT*a[i*3]);
        r[i*3 + 1] += dt*(v[i*3 + 1] + halfDT*a[i*3 + 1]);
        r[i*3 + 2] += dt*(v[i*3 + 2] + halfDT*a[i*3 + 2]);
            
        v[i*3] += halfDT * a[i*3];
        v[i*3 + 1] += halfDT * a[i*3 + 1];
        v[i*3 + 2] += halfDT * a[i*3 + 2];
    }

    //  Update accellerations from updated positions and potential
    computeAccelsAndPotential();

    //  Update velocity with updated acceleration
    for (int i=0; i<N; i++) {
        v[i*3] += halfDT*a[i*3];
        v[i*3 + 1] += halfDT*a[i*3 + 1];
        v[i*3 + 2] += halfDT*a[i*3 + 2];
    }
    
    // Elastic walls
    for (int i=0; i<N; i++) {
        for (int j=0; j<3; j++) {
            if (r[i*3 + j]<0.) {
                v[i*3 + j] *=-1.; //- elastic walls
                psum += 2*m*fabs(v[i*3 + j])/dt;  // contribution to pressure from "left" walls
            }
            if (r[i*3 + j]>=L) {
                v[i*3 + j]*=-1.;  //- elastic walls
                psum += 2*m*fabs(v[i*3 + j])/dt;  // contribution to pressure from "right" walls
            }
        }
    }
    
    return psum/(6*L*L);
}

void initializeVelocities() {
    
    for (int i=0; i<N; i++) {
        //  Pull a number from a Gaussian Distribution
        v[i*3] = gaussdist();
        v[i*3 + 1] = gaussdist();
        v[i*3 + 2] = gaussdist();
    }
    
    // Vcm = sum_i^N  m*v_i/  sum_i^N  M
    // Compute center-of-mas velocity according to the formula above
    double vCM[3] = {0, 0, 0};
    
    for (int i=0; i<N; i++) {
         vCM[0] += m*v[i*3];
         vCM[1] += m*v[i*3 + 1];
         vCM[2] += m*v[i*3 + 2];
    }
    
    
    double Nm = N*m;
    vCM[0] /= Nm;
    vCM[1] /= Nm;
    vCM[2] /= Nm;
    
    //  Subtract out the center-of-mass velocity from the
    //  velocity of each particle... effectively set the
    //  center of mass velocity to zero so that the system does
    //  not drift in space!
    for (int i=0; i<N; i++) {
        v[i*3] -= vCM[0];
        v[i*3 + 1] -= vCM[1];
        v[i*3 + 2] -= vCM[2];
    }
    
    //  Now we want to scale the average velocity of the system
    //  by a factor which is consistent with our initial temperature, Tinit
    double vSqdSum, lambda;
    vSqdSum=0.;
    for (int i=0; i<N; i++) {
        double vX = v[i*3];
        double vY = v[i*3 + 1];
        double vZ = v[i*3 + 2];

        vSqdSum += vX*vX + vY*vY + vZ*vZ;
    }
    
    lambda = sqrt( 3*(N-1)*Tinit/vSqdSum);
    
    for (int i=0; i<N; i++) {
        v[i*3] *= lambda;
        v[i*3 + 1] *= lambda;
        v[i*3 + 2] *= lambda;
    }
}

//  Numerical recipes Gaussian distribution number generator
double gaussdist() {
    static bool available = false;
    static double gset;
    double fac, rsq, v1, v2;
    if (!available) {
        do {
            v1 = 2.0 * rand() / double(RAND_MAX) - 1.0;
            v2 = 2.0 * rand() / double(RAND_MAX) - 1.0;
            rsq = v1 * v1 + v2 * v2;
        } while (rsq >= 1.0 || rsq == 0.0);
        
        fac = sqrt(-2.0 * log(rsq) / rsq);
        gset = v1 * fac;
        available = true;
        
        return v2*fac;
    } else {
        
        available = false;
        return gset;
        
    }
}
